#include "hip/hip_runtime.h"
// CUDA-01.cpp : Defines the entry point for the console application.
//

#include "stdafx.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <windows.h>
#include <math.h>
#include <hipfft/hipfft.h>

#define SIZE_ 1024 * 1024
#define N 75

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

static void HandleError(hipError_t err, const char *file, int line) {
	//hipError_t cudastatus = hipGetLastError();
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		getchar();
		exit(EXIT_FAILURE);
	}
}

__global__ void kernel(float *a, float *b, float *c) {
	c[threadIdx.x] =  a[threadIdx.x] + b[threadIdx.x];
	c[threadIdx.x] =  a[threadIdx.x] + b[threadIdx.x];
	c[threadIdx.x] =  a[threadIdx.x] + b[threadIdx.x];
	c[threadIdx.x] =  a[threadIdx.x] + b[threadIdx.x];
	c[threadIdx.x] =  a[threadIdx.x] + b[threadIdx.x];
	c[threadIdx.x] =  a[threadIdx.x] + b[threadIdx.x];
	c[threadIdx.x] =  a[threadIdx.x] + b[threadIdx.x];
	c[threadIdx.x] =  a[threadIdx.x] + b[threadIdx.x];
	c[threadIdx.x] =  a[threadIdx.x] + b[threadIdx.x];
	c[threadIdx.x] =  a[threadIdx.x] + b[threadIdx.x];
	c[threadIdx.x] =  a[threadIdx.x] + b[threadIdx.x];
	c[threadIdx.x] =  a[threadIdx.x] + b[threadIdx.x];
}

int _tmain(int argc, _TCHAR* argv[]) {
	size_t size = SIZE_ * sizeof(float);
	hipError_t e;

	e = hipSetDevice(0);
	HANDLE_ERROR(e);

	float *h, *hb, *hc;
	
	float *d, *db, *dc;
	
    e = hipMalloc(&d, size * N);
	HANDLE_ERROR(e);
    e = hipMalloc(&db, size * N);
	HANDLE_ERROR(e);
    e = hipMalloc(&dc, size * N);
	HANDLE_ERROR(e);

	//{
	//	h = (float *)malloc(size);
	//
	//	hipEvent_t start, stop;
	//
	//	hipEventCreate(&start);
	//	hipEventCreate(&stop);
	//
	//	hipEventRecord(start, 0);
	//	e = hipMemcpy(d, h, size, hipMemcpyHostToDevice);
	//	hipEventRecord(stop, 0);
	//	hipEventSynchronize(stop);
	//	HANDLE_ERROR(e);
	//
	//	float time;
	//	hipEventElapsedTime(&time, start, stop);
	//	printf("Czas dzialania ze str : %f\n", time);
	//
	//	free(h);
	//}

	{
		hipHostMalloc(&h, size * N);
		hipHostMalloc(&hb, size * N);
		hipHostMalloc(&hc, size * N);
		for (int i = 0; i < SIZE_ * N; i++)
			h[i] = 2;
		for (int i = 0; i < SIZE_ * N; i++)
			hb[i] = 3;

		hipEvent_t start, stop;
		hipStream_t stream1, stream2;
	
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipStreamCreate(&stream1);
		hipStreamCreate(&stream2);
		
		hipEventRecord(start, 0);
		for (int i = 0; i < SIZE_ * N; i += 2 * SIZE_)
		{
			e = hipMemcpyAsync(d, h + i, size, hipMemcpyHostToDevice, stream1);
			e = hipMemcpyAsync(d, h + i + SIZE_, size, hipMemcpyHostToDevice, stream2);
			
			e = hipMemcpyAsync(d, hb + i, size, hipMemcpyHostToDevice, stream1);
			e = hipMemcpyAsync(d, hb + i+ SIZE_, size, hipMemcpyHostToDevice, stream2);

			kernel<<< SIZE_ / 512, 512, 0, stream1 >>>(h, hb, hc);
			kernel<<< SIZE_ / 512, 512, 0, stream2 >>>(h, hb, hc);

			e = hipMemcpyAsync(h + i, d, size, hipMemcpyDeviceToHost, stream1);
			e = hipMemcpyAsync(h + i + SIZE_, d, size, hipMemcpyDeviceToHost, stream2);
		}
		hipStreamSynchronize(stream1);
		hipStreamSynchronize(stream2);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		HANDLE_ERROR(e);

		float time;
		hipEventElapsedTime(&time, start, stop);
		printf("Czas dzialania bez str: %f\n", time);

		hipHostFree(h);
		hipStreamDestroy(stream1);
		hipStreamDestroy(stream2);
	}

	hipFree(d);

	printf("Nacisnij Enter, aby zakonczyc.");
	getchar();
	return 0;
}

