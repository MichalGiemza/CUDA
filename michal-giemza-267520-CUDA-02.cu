#include "hip/hip_runtime.h"
// CUDA-01.cpp : Defines the entry point for the console application.
//

#include "stdafx.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>

#define N 32

__global__ void VecAdd(float *A, float *B, float *C) {
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}

void VecPrint(float *V, int n) {
	printf("[ ");
	for (int i = 0; i < n; i++) {
		printf("%2.0f ", V[i]);
	}
	printf("]\n");
}

int _tmain(int argc, _TCHAR* argv[]) {
    size_t size = N * sizeof(float);

	float *h_A, *h_B, *h_C;
	h_A = (float *)malloc(size);
	h_B = (float *)malloc(size);
	h_C = (float *)malloc(size);
	for (int i = 0; i < N; i++) {
		h_A[i] = i;
		h_B[i] = N - i;
	}

	float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
	
    VecAdd<<<1, N>>>(d_A, d_B, d_C);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
	
	printf("A: ");
	VecPrint(h_A, N);
	printf("\nB: ");
	VecPrint(h_B, N);
	printf("\nC: ");
	VecPrint(h_C, N);
	
	hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

	free(h_A);
	free(h_B);
	free(h_C);

	getchar();
	return 0;
}

