#include "hip/hip_runtime.h"
// CUDA-01.cpp : Defines the entry point for the console application.
//

#include "stdafx.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <windows.h>

#define N 64

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

static void HandleError(hipError_t err, const char *file, int line) {
	//hipError_t cudastatus = hipGetLastError();
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		getchar();
		exit(EXIT_FAILURE);
	}
}

static void MultMatCPU(float *A, float *B, float *M) {
	float sum, a, b;

	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			sum = 0;
			for (int k = 0; k < N; k++) {
				a = A[i * N + k];
				b = B[k * N + j];
				sum += a * b;
			}
			M[i * N + j] = sum;
		}
	}
}

__global__ void MultMat(float *A, float *B, float *M) {
	register float sum = 0;

	for (int k = 0; k < N; k++)
		sum += A[threadIdx.y * N + k] * B[k * N + threadIdx.x];

	M[threadIdx.y * N + threadIdx.x] = sum;
}

__global__ void MultBigMat(float *A, float *B, float *M) {
	register float sum = 0;

	for (int k = 0; k < N; k++)
		sum += A[blockIdx.x * N + k] * B[k * N + threadIdx.x];

	M[blockIdx.x * N + threadIdx.x] = sum;
}

__global__ void MultBigMat_3D(float *A, float *B, float *M) {
	__shared__ float mult[N];
	// Block x, y: M[x, y]
	// Thread x: k

	// Mnozenie
	mult[threadIdx.x] = A[blockIdx.y * N + threadIdx.x] * B[threadIdx.x * N + blockIdx.x];

	// Sumowanie
	for (int k = N / 2; k >= 1; k /= 2)
	{   // iteracje
		__syncthreads();
		if (threadIdx.x < k)
		{   // po tablicy
			mult[threadIdx.x] += mult[threadIdx.x + k];
		}
	}

	M[blockIdx.y * N + blockIdx.x] = mult[0];
}

void MatPrint(float *V, int n) {
	
	printf("\n(tylko pierwszy wiersz)");
	
	for (int i = 0; i < n; i++) {
		if (i % n == 0)
			putchar('\n');
		printf(" %5.1f", V[i]);
	}
	putchar('\n');
}

double getTime(LARGE_INTEGER start, LARGE_INTEGER end, LARGE_INTEGER countPerSec) {
	return (double)(end.QuadPart - start.QuadPart) / countPerSec.QuadPart * 1000;
}

int _tmain(int argc, _TCHAR* argv[]) {
    size_t size = N * N * sizeof(float);
	hipError_t e;

	float *h_A, *h_B, *h_M;

	h_A = (float *)malloc(size);
	h_B = (float *)malloc(size);
	h_M = (float *)malloc(size);

	for (int i = 0; i < N * N; i++) {
		h_A[i] = 1;
		h_B[i] = 1;
	}

	float *d_A, *d_B, *d_M;
    e = hipMalloc(&d_A, size);
    e = hipMalloc(&d_B, size);
    e = hipMalloc(&d_M, size);
	HANDLE_ERROR(e);

	e = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    e = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
	HANDLE_ERROR(e);
	
	//printf("Mnozenie macierzy %dx%d przez CPU:", N, N);
	//
	//MultMatCPU(h_A, h_B, h_M);
	//MatPrint(h_M, N); // Wypisanie
	
	printf("\nMnozenie macierzy %dx%d przez CUDA (Rozwiazanie z suma w petli):", N, N);

	dim3 block(N, 1, 1);
	dim3 grid(N, 1, 1);
	
    MultBigMat<<<grid, block>>>(d_A, d_B, d_M);
	hipDeviceSynchronize();

    e = hipMemcpy(h_M, d_M, size, hipMemcpyDeviceToHost);
	HANDLE_ERROR(e);

	MatPrint(h_M, N); // Wypisanie
	
	printf("\nMnozenie macierzy %dx%d przez CUDA (Rozwiazanie 3D):", N, N);

	dim3 grid3D(N, N, 1);
	
	MultBigMat_3D<<<grid3D, block>>>(d_A, d_B, d_M);
	hipDeviceSynchronize();

    e = hipMemcpy(h_M, d_M, size, hipMemcpyDeviceToHost);
	HANDLE_ERROR(e);

	MatPrint(h_M, N); // Wypisanie

	hipFree(d_A);
    hipFree(d_B);
    hipFree(d_M);

	free(h_A);
	free(h_B);
	free(h_M);

	getchar();
	return 0;
}

