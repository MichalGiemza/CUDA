#include "hip/hip_runtime.h"
// CUDA-01.cpp : Defines the entry point for the console application.
//

#include "stdafx.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <windows.h>

#define N 1024

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

static void HandleError(hipError_t err, const char *file, int line) {
	//hipError_t cudastatus = hipGetLastError();
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		getchar();
		exit(EXIT_FAILURE);
	}
}

__global__ void VecScalarProduct_0(float *A, float *B, float *SP) {
    //int x = blockIdx.x * blockDim.x + threadIdx.x;
    //int y = blockIdx.y * blockDim.y + threadIdx.y;
	register int i = threadIdx.x;

	float mult[N]; //?
	mult[i] = A[i] * B[i];

	__syncthreads();
	
	if (i == 0) {
		*SP = 0;
		for (int j = 0; j < N; j++)
			*SP += mult[j];
	}
}

__global__ void VecScalarProduct_SyncSum(float *A, float *B, float *SP) {
	register int i = threadIdx.x;

	__shared__ float mult[N];
	mult[i] = A[i] * B[i];

	__syncthreads();
	
	if (i == 0) {
		register float sum = 0;
		for (int j = 0; j < N; j++)
			sum += mult[j];

		*SP = sum;
	}
}

__global__ void VecScalarProduct_SyncSumNoReg(float *A, float *B, float *SP) {
	register int i = threadIdx.x;

	__shared__ float mult[N];
	mult[i] = A[i] * B[i];

	__syncthreads();
	
	if (i == 0) {
		*SP = 0;
		for (int j = 0; j < N; j++)
			*SP += mult[j];
	}
}

__global__ void VecScalarProduct(float *A, float *B, float *SP) {
	register int i = threadIdx.x;

	__shared__ float mult[N];
	mult[i] = A[i] * B[i];

	__syncthreads();
	
	for (int k = 2; k <= N; k *= 2)
	{   // iteracje
		__syncthreads();
		if (i % k == 0)
		{   // po tablicy
			mult[i] += mult[i + k/2];
		}
	}
	__syncthreads();
	*SP = mult[0];
}

__global__ void VecScalarProduct2(float *A, float *B, float *SP) {
	register int i = threadIdx.x;

	__shared__ float mult[N];
	mult[i] = A[i] * B[i];

	__syncthreads();
	
	for (int k = N / 2; k >= 1; k /= 2)
	{   // iteracje
		__syncthreads();
		if (i < k)
		{   // po tablicy
			mult[i] += mult[i + k];
		}
	}
	__syncthreads();
	*SP = mult[0];
}

void VecPrint(float *V, int n) {
	printf("[ ");
	for (int i = 0; i < n; i++) {
		printf("%2.0f ", V[i]);
	}
	printf("]\n");
}

double getTime(LARGE_INTEGER start, LARGE_INTEGER end, LARGE_INTEGER countPerSec) {
	return (double)(end.QuadPart - start.QuadPart) / countPerSec.QuadPart * 1000;
}

int _tmain(int argc, _TCHAR* argv[]) {
    size_t size = N * sizeof(float);
	hipError_t e;

	float *h_A, *h_B, h_SP;
	h_A = (float *)malloc(size);
	h_B = (float *)malloc(size);
	for (int i = 0; i < N; i++) {
		h_A[i] = 1;
		h_B[i] = 1;
	}

	float *d_A, *d_B, *d_SP;
    e = hipMalloc(&d_A, size);
    e = hipMalloc(&d_B, size);
    e = hipMalloc(&d_SP, sizeof(float));
	HANDLE_ERROR(e);

	e = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    e = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
	HANDLE_ERROR(e);
	
	dim3 block(N, 1, 1);
	dim3 grid(1, 1, 1);
	
	//LARGE_INTEGER countPerSec, timeA, timeB, timeC, timeD, timeE, timeF;
	//QueryPerformanceFrequency(&countPerSec);

	//QueryPerformanceCounter(&timeA);
    //VecScalarProduct_0<<<grid, block>>>(d_A, d_B, d_SP);
	//hipDeviceSynchronize();

	//QueryPerformanceCounter(&timeB);
    //VecScalarProduct_SyncSumNoReg<<<grid, block>>>(d_A, d_B, d_SP);
	//hipDeviceSynchronize();

	//QueryPerformanceCounter(&timeC);
    //VecScalarProduct_SyncSum<<<grid, block>>>(d_A, d_B, d_SP);
	//hipDeviceSynchronize();

	//QueryPerformanceCounter(&timeD);
    //VecScalarProduct<<<grid, block>>>(d_A, d_B, d_SP);
	//hipDeviceSynchronize();

	//QueryPerformanceCounter(&timeE);
    VecScalarProduct2<<<grid, block>>>(d_A, d_B, d_SP);
	//hipDeviceSynchronize();

	//QueryPerformanceCounter(&timeF);


	//printf("%fms - Czas dla wer. 0\n", getTime(timeA, timeB, countPerSec));
	//printf("%fms - Czas dla wer. bez rejestru\n", getTime(timeB, timeC, countPerSec));
	//printf("%fms - Czas dla wer. z suma synchr.\n", getTime(timeC, timeD, countPerSec));
	//printf("%fms - Czas dla wer. rownoleglej\n", getTime(timeD, timeE, countPerSec));
	//printf("%fms - Czas dla wer. zsunietej\n", getTime(timeE, timeF, countPerSec));

    e = hipMemcpy(&h_SP, d_SP, sizeof(float), hipMemcpyDeviceToHost);
	HANDLE_ERROR(e);

	printf("A = [1, 1, ..., 1], B = [1, 1, ..., 1]\n");
	printf("Iloczyn skalarny: %f", h_SP);
	
	hipFree(d_A);
    hipFree(d_B);
    hipFree(d_SP);

	free(h_A);
	free(h_B);

	getchar();
	return 0;
}

