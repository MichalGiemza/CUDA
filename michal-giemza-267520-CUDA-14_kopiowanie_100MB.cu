// CUDA-01.cpp : Defines the entry point for the console application.
//

#include "stdafx.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <windows.h>
#include <math.h>
#include <hipfft/hipfft.h>

#define INPUT_FILE "indeksyMedic.bin"
#define OUTPUT_FILE "wykresy_Z14.txt"
#define X 1024
#define Y 1024
#define N 100

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

static void HandleError(hipError_t err, const char *file, int line) {
	//hipError_t cudastatus = hipGetLastError();
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		getchar();
		exit(EXIT_FAILURE);
	}
}

int _tmain(int argc, _TCHAR* argv[]) {
	size_t size = X * Y * sizeof(float);
	hipError_t e;

	e = hipSetDevice(0);
	HANDLE_ERROR(e);

	float *h;
	
	float *d;
    e = hipMalloc(&d, size);
	HANDLE_ERROR(e);

	{
		h = (float *)malloc(size);

		hipEvent_t start, stop;
	
		hipEventCreate(&start);
		hipEventCreate(&stop);
	
		hipEventRecord(start, 0);
		e = hipMemcpy(d, h, size, hipMemcpyHostToDevice);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		HANDLE_ERROR(e);

		float time;
		hipEventElapsedTime(&time, start, stop);
		printf("Czas dzialania ze str : %f\n", time);

		free(h);
	}

	{
		hipHostMalloc(&h, size);

		hipEvent_t start, stop;
		hipStream_t stream;
	
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipStreamCreate(&stream);
	
		hipEventRecord(start, 0);
		hipMemcpyAsync(d, h, size, hipMemcpyHostToDevice, stream);
		hipStreamSynchronize(stream);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		HANDLE_ERROR(e);

		float time;
		hipEventElapsedTime(&time, start, stop);
		printf("Czas dzialania bez str: %f\n", time);

		hipHostFree(h);
		hipStreamDestroy(stream);
	}

	hipFree(d);

	printf("Nacisnij Enter, aby zakonczyc.");
	getchar();
	return 0;
}

